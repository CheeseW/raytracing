#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
//#include "vec3.h"
//#include "ray.h"
#include "hittable_list.h"
#include "sphere.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ": " << line << " '" << func << "'" << std::endl;
		hipDeviceReset();
		exit(99);
	}
}

__global__ void create_world(rayUtilities::Hittable** d_list, rayUtilities::Hittable** d_world ) {
    using namespace rayUtilities;

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new Sphere(Vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new Sphere(Vec3(0, -100.5, -1), 100);
        *d_world = new HittableList(d_list, 2);
    }
}

__device__ rayUtilities::Color ray_color(const rayUtilities::Ray& r, const rayUtilities::Hittable* world) {
    using namespace rayUtilities;

    HitRecord rec;
    if (world->hit(r, 0, FLT_MAX, rec)) return 0.5f*(rec.normal+Vec3(1,1,1));
    Vec3 d = r.direction().normalized();
    auto t = .5f * (d[1] + 1.f);
    const Color top{ .5,.7,1 };
    const Color bottom{ 1,1,1 };
    return t* top + (1.f - t) * bottom;
}

void write_image(const std::string filename, const rayUtilities::Color* fb, const int width, const int height) {
    using namespace rayUtilities;
    std::cout << "Writing image of size (" << width << ", " << height << ")" << std::endl;
    std::ofstream imgWritter(filename);
    imgWritter << "P3" << std::endl
        << width << " " << height << std::endl
        << "255" << std::endl;
    for (int j = height - 1; j >= 0; --j) 
        for (int i = 0; i < width; ++i) {
            int idx = j * width + i;
            int ir = static_cast<int>(255.999 * fb[idx][0]);
            int ig = static_cast<int>(255.999 * fb[idx][1]);
            int ib = static_cast<int>(255.999 * fb[idx][2]);

            imgWritter << ir << ' ' << ig << ' ' << ib << std::endl;
        }
    imgWritter.close();
    std::cout << "Done writing "<<filename << std::endl;
}

__global__ void render(rayUtilities::Color* fb, const rayUtilities::Hittable*const* world, int max_x, int max_y, const rayUtilities::Point3 lowerLeft, const rayUtilities::Vec3 horizontal, const rayUtilities::Vec3 vertical, const rayUtilities::Vec3 origin) {
    using namespace rayUtilities;

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < max_x && j < max_y) {
        const float u = float(i) / max_x;
        const float v = float(j) / max_y;
        const Ray ray(origin, lowerLeft + u * horizontal + v * vertical - origin);

        int idx = max_x * j + i;

        fb[idx] = ray_color(ray, *world);
    }
        
}

int main(int argc, char* argv[]) {

    using namespace rayUtilities;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Image
    const auto aspectRatio = 16. / 9;
    const int image_width = 800;
    const int image_height = static_cast<int>(image_width/aspectRatio);

    //camera
    const float viewport_height = 2.f;
    const float viewport_width = viewport_height*aspectRatio;
    const float focal_length = 1.f;

    const Point3 origin = Vec3(0, 0, 0);
    const Vec3 horizontal = Vec3(viewport_width, 0, 0);
    const Vec3 vertical = Vec3(0, viewport_height, 0);
    const Point3 lowerLeft = origin - (horizontal + vertical) / 2 - Vec3(0, 0, focal_length);

    // world
    Hittable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 3 * sizeof(Hittable*)));
    Hittable** d_world = d_list+2;
    create_world<<<1,1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    Color* fb;
   checkCudaErrors(hipMallocManaged((void**)&fb, image_width*image_height*sizeof(Color)));


   const int tx = 8;
   const int ty = 8;
   dim3 blocks((image_width + tx - 1) / tx, (image_height + ty - 1) / ty);
   dim3 threads(tx, ty);

   hipEventRecord(start);
   render << <blocks, threads >> > (fb, d_world, image_width, image_height, lowerLeft, horizontal, vertical, origin);
   hipEventRecord(stop);

   checkCudaErrors(hipGetLastError());
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipEventSynchronize(stop));

   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   std::cout << "Time to compute the frame buffer : " << milliseconds << " ms" << std::endl;

   write_image("image.ppm", fb, image_width, image_height);
  
   checkCudaErrors(hipFree(fb));
   return 0;
}
