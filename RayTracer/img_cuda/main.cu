#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "vec3.h"
#include "ray.h"


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ": " << line << " '" << func << "'" << std::endl;
		hipDeviceReset();
		exit(99);
	}
}

__device__ bool hit_sphere(const rayUtilities::Ray& r) {
    using namespace rayUtilities;
    const Point3 center(0, 0, -1);
    const float radius = .5;

    // Compute distance from center to ray
    const Vec3 oc = center - r.origin();
    const auto& d = r.direction();

    double a = d.dot(d);
    double b = -2 * d.dot(oc);
    double c = oc.dot(oc) - radius * radius;
    double discriminant = b * b - 4 * a * c;
    return discriminant >= 0;

}

__device__ rayUtilities::Color ray_color(const rayUtilities::Ray& r) {
    using namespace rayUtilities;
    if (hit_sphere(r))
        return Color(1, 0, 0);
    Vec3 d = r.direction().normalized();
    auto t = .5f * (d[1] + 1.f);
    const Color top{ .5,.7,1 };
    const Color bottom{ 1,1,1 };
    return t* top + (1.f - t) * bottom;
}
#if 1

void write_image(const std::string filename, const rayUtilities::Color* fb, const int width, const int height) {
    using namespace rayUtilities;
    std::cout << "Writing image of size (" << width << ", " << height << ")" << std::endl;
    std::ofstream imgWritter(filename);
    imgWritter << "P3" << std::endl
        << width << " " << height << std::endl
        << "255" << std::endl;
    for (int j = height - 1; j >= 0; --j) 
        for (int i = 0; i < width; ++i) {
            int idx = j * width + i;
            int ir = static_cast<int>(255.999 * fb[idx][0]);
            int ig = static_cast<int>(255.999 * fb[idx][1]);
            int ib = static_cast<int>(255.999 * fb[idx][2]);

            imgWritter << ir << ' ' << ig << ' ' << ib << std::endl;
        }
    imgWritter.close();
    std::cout << "Done writing "<<filename << std::endl;
}

#endif
#if 1
__global__ void render(rayUtilities::Color* fb, int max_x, int max_y, const rayUtilities::Point3 lowerLeft, const rayUtilities::Vec3 horizontal, const rayUtilities::Vec3 vertical, const rayUtilities::Vec3 origin) {
    using namespace rayUtilities;

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < max_x && j < max_y) {
        const float u = float(i) / max_x;
        const float v = float(j) / max_y;
        const Ray ray(origin, lowerLeft + u * horizontal + v * vertical - origin);

        int idx = max_x * j + i;

        fb[idx] = ray_color(ray);
       
    }
        
}
#endif

int main(int argc, char* argv[]) {
    using namespace rayUtilities;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Image
    const auto aspectRatio = 16. / 9;
    const int image_width = 800;
    const int image_height = static_cast<int>(image_width/aspectRatio);

    //camera
    const float viewport_height = 2.f;
    const float viewport_width = viewport_height*aspectRatio;
    const float focal_length = 1.f;

    const Point3 origin = Vec3(0, 0, 0);
    const Vec3 horizontal = Vec3(viewport_width, 0, 0);
    const Vec3 vertical = Vec3(0, viewport_height, 0);
    const Point3 lowerLeft = origin - (horizontal + vertical) / 2 - Vec3(0, 0, focal_length);
#if 1
    Color* fb;
   checkCudaErrors(hipMallocManaged((void**)&fb, image_width*image_height*sizeof(Color)));

   const int tx = 8;
   const int ty = 8;
   dim3 blocks((image_width + tx - 1) / tx, (image_height + ty - 1) / ty);
   dim3 threads(tx, ty);

   hipEventRecord(start);
   render << <blocks, threads >> > (fb, image_width, image_height, lowerLeft, horizontal, vertical, origin);
   hipEventRecord(stop);

   checkCudaErrors(hipGetLastError());
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipEventSynchronize(stop));

   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   std::cout << "Time to compute the frame buffer : " << milliseconds << " ms" << std::endl;

   write_image("image.ppm", fb, image_width, image_height);
  
   checkCudaErrors(hipFree(fb));
#endif
   return 0;
}
